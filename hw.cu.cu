#include <stdio.h>
#include <hip/hip_runtime.h>
float element1;
float cputogpu1;
float kernel1;
float gputocpu1;
float element2;
float cputogpu2;
float kernel2;
float gputocpu2;
float element3;
float cputogpu3;
float kernel3;
float gputocpu3;
float element4;
float cputogpu4 = 0;
float kernel4 = 0;
float gputocpu4 = 0;
__global__ void arradd( float *A, int N)
{
int  B = 2000;
int  i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < N)
{
A[i] = A[i] + B;
}

}

__global__ void darradd( double *A2, int N2)
{
int B2 = 2000;
int i2 = blockDim.x * blockIdx.x + threadIdx.x;
if (i2 < N2)
{
A2[i2] = A2[i2] + B2;
}
}

__global__ void iarradd( int32_t *A3, int N3)
{
int B3 = 2000;
int i3 = blockDim.x * blockIdx.x + threadIdx.x;
if (i3 < N3)
{
A3[i3] = A3[i3] + B3;
}
}

__global__ void xarradd( float *A4, int N4, int B4, int num)
{
int i4 = blockDim.x * blockIdx.x + threadIdx.x;
if (i4 < N4)
{
for (int i=0; i<num; i++)
{
A4[i4] = A4[i4] + B4;
}
}
}

int helper4(int num){
hipError_t err4 = hipSuccess;
hipEvent_t start41, stop41;
hipEvent_t start42, stop42;
hipEvent_t start43, stop43;
float time41;
float time42;
float time43;
int N4 = 128000000;
int B4 = 2000;
size_t size4 = N4 *sizeof(float);

float *h_A4 = (float *)malloc(size4);


//float *h_C4 = (float *)malloc(size4);


for (int i4 = 0; i4 < N4; i4++)
{
h_A4[i4] = i4/3.0f;
}

float *d_A4 = NULL;
err4 = hipMalloc((void **)&d_A4, size4);

//float *d_C4 = NULL;
//err4 = cudaMalloc((void **)&d_C4, size4);


hipEventCreate(&start41);
hipEventRecord(start41,0);

//printf("COPY input data from the host to CUDA device\n");
err4 = hipMemcpy(d_A4, h_A4, size4, hipMemcpyHostToDevice);

hipEventCreate(&stop41);
hipEventRecord(stop41,0);
hipEventSynchronize(stop41);
hipEventElapsedTime(&time41, start41, stop41);
//printf("The time for CPU to GPU is %fms\n",time41);
cputogpu4 = time41;

hipEventCreate(&start42);
hipEventRecord(start42,0);

int threadsPerBlock = 256;
int blocksPerGrid = (N4 + threadsPerBlock - 1) / threadsPerBlock;
//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
xarradd<<<blocksPerGrid, threadsPerBlock>>>(d_A4, N4, B4, num);
err4 = hipGetLastError();

/*if (err != cudaSuccess)
{
printf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", cudaGetErrorString(err));
exit(EXIT_FAILURE);
*/
hipEventCreate(&stop42);
hipEventRecord(stop42,0);
hipEventSynchronize(stop42);
hipEventElapsedTime(&time42,start42,stop42);
//printf("The time for kernal is %fms\n",time42);
kernel4 = time42;
hipEventCreate(&start43);
hipEventRecord(start43,0);

//printf("Copy output data from the CUDA device to the host memory\n");

err4 = hipMemcpy(h_A4, d_A4, size4, hipMemcpyDeviceToHost);

hipEventCreate(&stop43);
hipEventRecord(stop43,0);
hipEventSynchronize(stop43);
hipEventElapsedTime(&time43,start43,stop43);
//printf("The time for GPU to CPU is %fms\n",time43);
gputocpu4 = time43;


err4 = hipFree(d_A4);
//err4 = cudaFree(d_C4);


free(h_A4);
//free(h_C4);

err4 = hipDeviceReset();


//printf("test done\n");
//printf("Done\n");
return 0;

}

int helper3(int N3){
hipError_t err3 = hipSuccess;
hipEvent_t start31, stop31;
hipEvent_t start32, stop32;
hipEvent_t start33, stop33;
float time31;
float time32;
float time33;
N3=N3*1000000;
size_t size3 = N3 *sizeof(int32_t);

int32_t *h_A3 = (int32_t *)malloc(size3);


//float *h_C3 = (float *)malloc(size3);


for (int i = 0; i < N3; i++)
{
h_A3[i] = i/3.0f;
}

int32_t *d_A3 = NULL;
err3 = hipMalloc((void **)&d_A3, size3);

//float *d_C3 = NULL;
//err3 = cudaMalloc((void **)&d_C3, size3);


hipEventCreate(&start31);
hipEventRecord(start31,0);

//printf("COPY input data from the host to CUDA device\n");
err3 = hipMemcpy(d_A3, h_A3, size3, hipMemcpyHostToDevice);

hipEventCreate(&stop31);
hipEventRecord(stop31,0);
hipEventSynchronize(stop31);
hipEventElapsedTime(&time31, start31, stop31);
//printf("The time for CPU to GPU is %fms\n",time31);
cputogpu3 = time31;

hipEventCreate(&start32);
hipEventRecord(start32,0);

int threadsPerBlock = 256;
int blocksPerGrid = (N3 + threadsPerBlock - 1)/threadsPerBlock;
//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
iarradd<<<blocksPerGrid, threadsPerBlock>>>(d_A3, N3);
err3 = hipGetLastError();

/*if (err != cudaSuccess)
{
printf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", cudaGetErrorString(err));
exit(EXIT_FAILURE);
}*/
hipEventCreate(&stop32);
hipEventRecord(stop32,0);
hipEventSynchronize(stop32);
hipEventElapsedTime(&time32,start32,stop32);
//printf("The time for kernal is %fms\n",time32);
kernel3 = time32;
hipEventCreate(&start33);
hipEventRecord(start33,0);

//printf("Copy output data from the CUDA device to the host memory\n");


err3 = hipMemcpy(h_A3, d_A3, size3, hipMemcpyDeviceToHost);

if (err3 != hipSuccess)
{
fprintf(stderr, "Failed to copy vector c from device to host (error code %s)!\n", hipGetErrorString(err3));
exit(EXIT_FAILURE);
}

hipEventCreate(&stop33);
hipEventRecord(stop33,0);
hipEventSynchronize(stop33);
hipEventElapsedTime(&time33,start33,stop33);
//printf("The time for GPU to CPU is %fms\n",time33);
gputocpu3 = time33;


err3 = hipFree(d_A3);
//err3 = cudaFree(d_C3);


free(h_A3);
//free(h_C3);

err3 = hipDeviceReset();


//printf("test done\n");
//printf("Done\n");
return 0;

}


int helper2(int N2) {
hipError_t err2 = hipSuccess;
hipEvent_t start21, stop21;
hipEvent_t start22, stop22;
hipEvent_t start23, stop23;
float time21;
float time22;
float time23;
N2 = N2*1000000;

size_t size2 = N2 *sizeof(double);

double *h_A2 = (double *)malloc(size2);


//float *h_C2 = (float *)malloc(size2);


for (int i2 = 0; i2 < N2; i2++)
{
h_A2[i2] = i2/3.0f;
}

double *d_A2 = NULL;
err2 = hipMalloc((void **)&d_A2, size2);

//float *d_C2 = NULL;
//err2 = cudaMalloc((void **)&d_C2, size2);


hipEventCreate(&start21);
hipEventRecord(start21,0);

//printf("COPY input data from the host to CUDA device\n");
err2 = hipMemcpy(d_A2, h_A2, size2, hipMemcpyHostToDevice);

hipEventCreate(&stop21);
hipEventRecord(stop21,0);
hipEventSynchronize(stop21);
hipEventElapsedTime(&time21, start21, stop21);
//printf("The time for CPU to GPU is %fms\n",time21);
cputogpu2 = time21;
hipEventCreate(&start22);
hipEventRecord(start22,0);

int threadsPerBlock = 256;
int blocksPerGrid = (N2 + threadsPerBlock - 1) / threadsPerBlock;
//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
darradd<<<blocksPerGrid, threadsPerBlock>>>(d_A2, N2);
err2 = hipGetLastError();

/*if (err != cudaSuccess)
{
printf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", cudaGetErrorString(err));
exit(EXIT_FAILURE);
}*/
hipEventCreate(&stop22);
hipEventRecord(stop22,0);
hipEventSynchronize(stop22);
hipEventElapsedTime(&time22,start22,stop22);
//printf("The time for kernal is %fms\n",time22);
kernel2 = time22;


hipEventCreate(&start23);
hipEventRecord(start23,0);

//printf("Copy output data from the CUDA device to the host memory\n");
err2 = hipMemcpy(h_A2, d_A2, size2, hipMemcpyDeviceToHost);

if (err2 != hipSuccess)
{
fprintf(stderr, "Failed to copy vector c from device to host (error code %s)!\n", hipGetErrorString(err2));
exit(EXIT_FAILURE);
}

hipEventCreate(&stop23);
hipEventRecord(stop23,0);
hipEventSynchronize(stop23);
hipEventElapsedTime(&time23,start23,stop23);
//printf("The time for GPU to CPU is %fms\n",time23);
gputocpu2=time23;


err2 = hipFree(d_A2);
//err2 = cudaFree(d_C2);


free(h_A2);
//free(h_C2);

err2 = hipDeviceReset();


//printf("test done\n");
//printf("Done\n");
return 0;

}




int helper(int N){
hipError_t err = hipSuccess;
hipEvent_t start1, stop1;
hipEvent_t start2, stop2;
hipEvent_t start3, stop3;
float time1;
float time2;
float time3;
N = N * 1000000;


size_t size = N *sizeof(float);

float *h_A = (float *)malloc(size);


//float *h_C = (float *)malloc(size);


for (int i = 0; i < N; i++)
{
h_A[i] = i/3.0f;
}

float *d_A = NULL;
err = hipMalloc((void **)&d_A, size);

//float *d_C = NULL;
//err = cudaMalloc((void **)&d_C, size);


hipEventCreate(&start1);
hipEventRecord(start1,0);

//printf("COPY input data from the host to CUDA device\n");
err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

hipEventCreate(&stop1);
hipEventRecord(stop1,0);
hipEventSynchronize(stop1);
hipEventElapsedTime(&time1, start1, stop1);
//printf("The time for CPU to GPU is %fms\n",time1);
cputogpu1 = time1;

hipEventCreate(&start2);
hipEventRecord(start2,0);

int threadsPerBlock = 256;
int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
arradd<<<blocksPerGrid, threadsPerBlock>>>(d_A, N);
err = hipGetLastError();

/*if (err != cudaSuccess)
{
printf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", cudaGetErrorString(err));
exit(EXIT_FAILURE);
}*/
hipEventCreate(&stop2);
hipEventRecord(stop2,0);
hipEventSynchronize(stop2);
hipEventElapsedTime(&time2,start2,stop2);
//printf("The time for kernal is %fms\n",time2);
kernel1 = time2;

hipEventCreate(&start3);
hipEventRecord(start3,0);

//printf("Copy output data from the CUDA device to the host memory\n");
err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);

/*if (err != cudaSuccess)
{
fprintf(stderr, "Failed to copy vector c from device to host (error code %s)!\n", cudaGetErrorString(err));
exit(EXIT_FAILURE);
}*/

hipEventCreate(&stop3);
hipEventRecord(stop3,0);
hipEventSynchronize(stop3);
hipEventElapsedTime(&time3,start3,stop3);
//printf("The time for GPU to CPU is %fms\n",time3);
gputocpu1 = time3;


err = hipFree(d_A);
//err = cudaFree(d_C);


free(h_A);
//free(h_C);

err = hipDeviceReset();


//printf("test done\n");
//printf("Done\n");
return 0;

}

int main(void){

int a[9] = {1, 2, 4, 8, 16, 32, 64, 128, 256};
printf("part A\n");
printf("Elements    CPUtoGPU(ms)    Kernel (ms)    GPUtoCPU (ms)\n");
for (int i=0; i<=8;i++){
helper(a[i]);
element1 = a[i];
printf("%6f   ", element1);
printf("%11f   ", cputogpu1);
printf("%15f   ", kernel1);
printf("%12f  \n ", gputocpu1);
}
int b[9] = {1, 2, 4, 8, 16, 32, 64, 128, 256};
printf("part B\n");
printf("Elements    CPUtoGPU(ms)    Kernel (ms)    GPUtoCPU (ms)\n");
for (int i2=0; i2<=8;i2++){
helper2(b[i2]);
element2 = b[i2];
printf("%6f   ", element2);
printf("%11f   ", cputogpu2);
printf("%15f   ", kernel2);
printf("%12f  \n ", gputocpu2);

}
printf("part C\n");
printf("Elements    CPUtoGPU(ms)    Kernel (ms)    GPUtoCPU (ms)\n");

int c[9] = {1, 2, 4, 8, 16, 32, 64, 128, 256};
for (int i3=0; i3<=8;i3++){
helper3(c[i3]);
element3 = b[i3];
printf("%6f   ", element3);
printf("%11f   ", cputogpu3);
printf("%15f   ", kernel3);
printf("%12f  \n ", gputocpu3);


}

printf("part D\n");
printf("XaddedTimes    CPUtoGPU(ms)    Kernel (ms)    GPUtoCPU (ms)     Elements (m)\n");
int x4 = 128;
int d[9] = {1, 2, 4, 8, 16, 32, 64, 128, 256};
for (int i4=0; i4<=8;i4++){
helper4(d[i4]);
element4 = d[i4];
printf("%6f   ", element4);
printf("%12f   ", cputogpu4);
printf("%16f   ", kernel4);
printf("%13f   ", gputocpu4);
printf("%13d  \n", x4);


}
}
    