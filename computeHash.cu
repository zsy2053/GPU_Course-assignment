#include "hip/hip_runtime.h"
/*$Id: computeHash.cu 2016-02-05 19:42:18 (author: Reza Mokhtari)$*/
#include <stdio.h>
#include <unistd.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <time.h>
#include <sys/time.h>

__global__ void hashKernel(char* input, int size, int* indices, int* hashOutput)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x; /* assuming 1D grid and block */

	//Each thread hashes the data from indices[index] to indices[index + 1]
	int start = indices[index];
	int end = indices[index + 1];
	
	unsigned hash = 2166136261;
	unsigned FNVMultiple = 16777619;

	for(int i = start; i < end; i += sizeof(int))
	{
		int arrayValue = *((int*) (input + i));
		hash += arrayValue;
                hash = hash ^ (arrayValue);     /* xor  the entire 32 bits */
                hash -= arrayValue;
                hash = hash * FNVMultiple;  /* multiply by the magic number */
                hash *= (arrayValue == 0)? 1 : arrayValue;
	}

	hashOutput[index] = hash;
}

int main(int argc, char** argv)
{

	int fd;
        char *fdata;
        struct stat finfo;
        char *fname;

        if (argc < 2)
        {
                printf("USAGE: %s <inputfilename>\n", argv[0]);
                exit(1);
        }

        fname = argv[1];
        fd = open(fname, O_RDONLY);
        fstat(fd, &finfo);

        printf("Allocating %lluMB for the input file.\n", ((long long unsigned int)finfo.st_size) / (1 << 20));
        fdata = (char *) malloc(finfo.st_size);
        size_t successRead = read (fd, fdata, finfo.st_size);
        size_t fileSize = (size_t) finfo.st_size;

        if(successRead != fileSize)
	{
                printf("Not all of the file is read, terminating...\n"); /* happens when input data is too large. Not going to handle this for now */
		exit(1);
	}

	//setting fixed number of threads, do not modify.
	dim3 grid(8, 1, 1);
	dim3 block(512, 1, 1);
	int numThreads = grid.x * block.x;

	int* indices = (int*) malloc((numThreads + 1) * sizeof(int));
	
	//calculating indices. Each index shows the point from which a thread starts hashing the input data
	int inputChunkSize = (fileSize + (numThreads - 1)) / numThreads;
        
        inputChunkSize = inputChunkSize - (inputChunkSize % 4);
        
	for(int i = 0; i < numThreads ; i ++)
		indices[i] = i * inputChunkSize; /* last thread(s) might go out of boundary, but gonna be handled in the kernel */
	//Setting the (last + 1) index
	indices[numThreads] = (int) fileSize;

	int* d_indices;
	hipMalloc((void**) &d_indices, numThreads * sizeof(int));
	hipMemcpy(d_indices, indices, numThreads * sizeof(int), hipMemcpyHostToDevice);

	char* d_input;
	hipMalloc((void**) &d_input, fileSize);
	hipMemcpy(d_input, fdata, fileSize, hipMemcpyHostToDevice);

	//Each thread will store its hash value in an element of this array.
	int* d_hashOutput;
	hipMalloc((void**) &d_hashOutput, numThreads * sizeof(int));
	hipMemset(d_hashOutput, 0, numThreads * sizeof(int));

	struct timeval partial_start, partial_end;
        time_t sec, ms, diff;
        gettimeofday(&partial_start, NULL);

	hashKernel<<<grid, block>>>(d_input, fileSize, d_indices, d_hashOutput);
	hipDeviceSynchronize();
	
        hipError_t errR = hipGetLastError();
        if(errR != hipSuccess)
	{
		printf("Kernel returned an error, terminating...\n");
                exit(1);
	}

	gettimeofday(&partial_end, NULL);
        sec = partial_end.tv_sec - partial_start.tv_sec;
        ms = partial_end.tv_usec - partial_start.tv_usec;
        diff = sec * 1000000 + ms;

        printf("\n%10s:\t\t%0.1fms\n", "Kernel elapsed time", (double)((double)diff/1000.0));
	int* hashOutput = (int*) malloc(numThreads * sizeof(int));
	hipMemcpy(hashOutput, d_hashOutput, numThreads * sizeof(int), hipMemcpyDeviceToHost);

	//Summing up the threads' hash values to form one final hash value.
	int finalHashValue = 0;
	for(int i = 0; i < numThreads; i ++)
		finalHashValue += hashOutput[i];
	
	printf("Final hash value: %d\n", finalHashValue);
	
	return 0;


}
