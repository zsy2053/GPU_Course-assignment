#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <ctype.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <time.h>
#include <sys/time.h>


#include "imageFilter_kernel.cu"

#define IMG_DATA_OFFSET_POS 10
#define BITS_PER_PIXEL_POS 28

int swap;
void test_endianess();
void swap_bytes(char *bytes, int num_bytes);

int main(int argc, char *argv[]) 
{
	int i;
	int fd;
	char *fdata;
	struct stat finfo;
	char * inputfname;
	char * outputfname;

	if (argc < 4)
	{
		printf("USAGE: %s <bitmap input filename> <bitmap output file name> <part specifier>\n", argv[0]);
		exit(1);
	}

	inputfname = argv[1];
	outputfname = argv[2];
	char partId = argv[3][0];
	if(partId != 'a' && partId != 'b' && partId != 'c')
	{
		printf("Please provide a part specifier: a, b, or c\n");
		exit(1);
	}

	printf("Image filter: Running...\n");

	fd = open(inputfname, O_RDONLY);
	fstat(fd, &finfo);

	fdata = (char*) malloc(finfo.st_size);
	
	read (fd, fdata, finfo.st_size);

	if ((fdata[0] != 'B') || (fdata[1] != 'M')) 
	{
		printf("File is not a valid bitmap file. Terminating the program\n");
		exit(1);
	}

	test_endianess();     // will set the variable "swap"

	unsigned short *bitsperpixel = (unsigned short *)(&(fdata[BITS_PER_PIXEL_POS]));
	if (swap) 
	{
		printf("swapping\n");
		swap_bytes((char *)(bitsperpixel), sizeof(*bitsperpixel));
	}

 	// ensure its 3 bytes per pixel
	if (*bitsperpixel != 24) 
	{
		printf("Error: Invalid bitmap format - ");
		printf("This application only accepts 24-bit pictures. Exiting\n");
		exit(1);
	}

	unsigned short *data_pos = (unsigned short *)(&(fdata[IMG_DATA_OFFSET_POS]));
	if (swap) 
	{
		swap_bytes((char *)(data_pos), sizeof(*data_pos));
	}

	int imgdata_bytes = (int)finfo.st_size - (int)(*(data_pos));
	printf("This file has %d bytes of image data, %d pixels\n", imgdata_bytes, imgdata_bytes / 3);

	int width = *((int*)&fdata[18]);
	printf("Width: %d\n", width);
	int height = *((int*)&fdata[22]);
	printf("Height: %d\n", height);

	int fileSize = (int) finfo.st_size;	

	//p will point to the first pixel
	char* p = &(fdata[*data_pos]);

	//Set the number of blocks and threads
	dim3 grid(12, 1, 1);
	dim3 block(1024, 1, 1);

	char* d_inputPixels;
	hipMalloc((void**) &d_inputPixels, width * height * 3);
	hipMemcpy(d_inputPixels, p, width * height * 3, hipMemcpyHostToDevice);
	
	char* d_outputPixels;
	hipMalloc((void**) &d_outputPixels, width * height * 3);
	hipMemset(d_outputPixels, 0, width * height * 3);

	struct timeval start_tv, end_tv;
	time_t sec;
	time_t ms;
	time_t diff;
	gettimeofday(&start_tv, NULL);

    int numberOfthread = grid.x * block.x;
    int pixelPerthread = (width * height) / numberOfthread;
	if((width * height) % numberOfthread != 0)
        pixelPerthread += 1;
	

    
	int blocksRows = (width - 8) / 120;
	int blocksCols = (height - 8) / 120;
	if((width - 8) % 120 != 0)
		blocksRows += 1;

	if((height - 8) % 120 != 0) 
		blocksCols += 1;
    
    int times = (blocksRows * blocksCols) / 12;
    if((blocksRows * blocksCols) % 12 != 0)
        times += 1;

	if(partId == 'a')
	{
		imageFilterKernelPartA<<<grid, block>>>((char3*) d_inputPixels, (char3*) d_outputPixels, width, height , pixelPerthread);
	} 
	else if(partId == 'b')
	{
		imageFilterKernelPartB<<<grid, block>>>((char3*) d_inputPixels, (char3*) d_outputPixels, width, height , pixelPerthread, numberOfthread);
	}
	else if(partId == 'c')
	{
		imageFilterKernelPartC<<<grid, block>>>((char3*) d_inputPixels, (char3*) d_outputPixels, width, height, blocksRows, blocksCols, times);
	}

	hipDeviceSynchronize();

	gettimeofday(&end_tv, NULL);
	sec = end_tv.tv_sec - start_tv.tv_sec;
	ms = end_tv.tv_usec - start_tv.tv_usec;

	diff = sec * 1000000 + ms;

	printf("%10s:\t\t%fms\n", "Time elapsed", (double)((double)diff/1000.0));


	char* outputPixels = (char*) malloc(height * width * 3);
	hipMemcpy(outputPixels, d_outputPixels, height * width * 3, hipMemcpyDeviceToHost);

	memcpy(&(fdata[*data_pos]), outputPixels, height * width * 3);

	FILE *writeFile; 
	writeFile = fopen(outputfname,"w+");
	for(i = 0; i < fileSize; i++)
		fprintf(writeFile,"%c", fdata[i]);
	fclose(writeFile);

	return 0;
} 

void test_endianess() {
    unsigned int num = 0x12345678;
    char *low = (char *)(&(num));
    if (*low ==  0x78) {
        //dprintf("No need to swap\n");
        swap = 0;
    }
    else if (*low == 0x12) {
        //dprintf("Need to swap\n");
        swap = 1;
    }
    else {
        printf("Error: Invalid value found in memory\n");
        exit(1);
    } 
}

void swap_bytes(char *bytes, int num_bytes) 
{
    int i;
    char tmp;
    
    for (i = 0; i < num_bytes/2; i++) {
        //dprintf("Swapping %d and %d\n", bytes[i], bytes[num_bytes - i - 1]);
        tmp = bytes[i];
        bytes[i] = bytes[num_bytes - i - 1];
        bytes[num_bytes - i - 1] = tmp;    
    }
}
