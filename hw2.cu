#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <omp.h>
	
#define THREADS_PER_BLOCK	512
#define BLOCKS_PER_GRID_ROW 128
float cpu1;
float cpu2;
float gpu1;
float gpu2;
float max1;
float min1;
float m1;
float m2;
float cc1;
float cc2;
float cc3;

__global__ void arradd( float *A)
{

__shared__ float max[512];

int arrayIndex = 128*512*blockIdx.y + 512*blockIdx.x + threadIdx.x;
max[threadIdx.x] = A[arrayIndex];

__syncthreads();
int nTotalThreads = blockDim.x;

while(nTotalThreads > 1)
	{
		int halfPoint = (nTotalThreads >> 1);	
		if (threadIdx.x < halfPoint)
		{
                        float temp = max[threadIdx.x + halfPoint];
			if (temp > max[threadIdx.x]) max[threadIdx.x] = temp;
		}
		__syncthreads();

		nTotalThreads = (nTotalThreads >> 1);	// divide by two.
	}
if (threadIdx.x == 0)
	{
		A[128*blockIdx.y + blockIdx.x] = max[0];
	}
}



__global__ void erredd( float *A)
{
__shared__ float min[512];

int arrayIndex = 128*512*blockIdx.y + 512*blockIdx.x + threadIdx.x;
min[threadIdx.x] = A[arrayIndex];


__syncthreads();
int nTotalThreads = blockDim.x;

while(nTotalThreads > 1)
{
  int halfPoint = (nTotalThreads >> 1);
  if(threadIdx.x < halfPoint)
 {
 float temp = min[threadIdx.x + halfPoint];
 if (temp < min[threadIdx.x]) min[threadIdx.x] = temp;
}
__syncthreads();

nTotalThreads = (nTotalThreads >> 1);
}
if (threadIdx.x == 0)
{
A[128*blockIdx.y + blockIdx.x] = min[0];
}

}

void helper(float *A, int N){
hipEvent_t start2, stop2;

float time1;


 
if (N <=0) return;
float max;
max = A[0];
for (int i=0; i<10; i++)
{
hipEventCreate(&start2);
hipEventRecord(start2,0);
for (int i=1; i<N; i++)
{
float temp = A[i];
if (temp > max) max = temp;
}
hipEventCreate(&stop2);
hipEventRecord(stop2,0);
hipEventSynchronize(stop2);
hipEventElapsedTime(&time1, start2, stop2);
time1 = time1 + time1;
}
            cpu1 = time1 / 10;
            cpu1 = cpu1 / 1000;
                           
		max1=max;
}

void helper2(float *B, int N){
hipEvent_t start3, stop3;
float time2;



if (N <=0) return;
float min;
min = B[0];
for (int i=0; i<10; i++)
{
hipEventCreate(&start3);
hipEventRecord(start3,0);
for (int i=1; i<N; i++)
{
float temp = B[i];
if (temp < min) min = temp;
}
hipEventCreate(&stop3);
hipEventRecord(stop3,0);
hipEventSynchronize(stop3);
hipEventElapsedTime(&time2, start3, stop3);
time2 = time2 + time2;
}
		cpu2 = time2 / 10;
                cpu2 = cpu2 / 1000;
		min1=min;
}





void step1Max(int N){
hipEvent_t start2, stop2;
hipEvent_t start21, stop21;
hipEvent_t start22, stop22;
float time22;
float time2;
float time29;
float time21;
N = N * 1048576;
        
   	float *d_A;  
     size_t size = N *sizeof(float);
     float *h_A = (float *)malloc(size);


	hipMalloc((void **)&d_A, sizeof(float) * N);
	
	for(int i = 0; i < N; i++)
	{
		h_A[i] = (float)rand();
	}

		
	float tempMax;
                               
              
		

		         int blockGridWidth = BLOCKS_PER_GRID_ROW;
		          int blockGridHeight = (N / THREADS_PER_BLOCK) / blockGridWidth;

		         dim3 blockGridRows(blockGridWidth, blockGridHeight);
		         dim3 threadBlockRows(THREADS_PER_BLOCK, 1);

                        int k=0;
			while (k!=10)
			{		
                        hipMemcpy(d_A, h_A, sizeof(float) * N, hipMemcpyHostToDevice);
                        hipEventCreate(&start2);
                        hipEventRecord(start2,0);
			arradd<<<blockGridRows, threadBlockRows>>>(d_A);
			hipEventCreate(&stop2);
                        hipEventRecord(stop2,0);
                        hipEventSynchronize(stop2);
                        hipEventElapsedTime(&time2, start2, stop2);
			hipDeviceSynchronize();
			hipMemcpy(h_A, d_A, sizeof(float) * N / THREADS_PER_BLOCK, hipMemcpyDeviceToHost);
			tempMax = h_A[0];
			for (int i = N / THREADS_PER_BLOCK; i > 0; i = i / 2)
			{
			hipMemcpy(d_A, h_A, sizeof(float) * i, hipMemcpyHostToDevice);
                        hipEventCreate(&start21);
                        hipEventRecord(start21,0);
			arradd<<<blockGridRows, threadBlockRows>>>(d_A);
			hipEventCreate(&stop21);
                        hipEventRecord(stop21,0);
                        hipEventSynchronize(stop21);
                        hipEventElapsedTime(&time21, start21, stop21);
			time21 = time21 + time21;
			hipDeviceSynchronize();
			hipMemcpy(h_A, d_A, sizeof(float) * i, hipMemcpyDeviceToHost);
			tempMax = h_A[0];
			if (i==1)
			{
                        hipMemcpy(d_A, h_A, sizeof(int) * THREADS_PER_BLOCK, hipMemcpyHostToDevice);
                        hipEventCreate(&start22);
                        hipEventRecord(start22,0);
			arradd<<<blockGridRows, threadBlockRows>>>(d_A);
			hipEventCreate(&stop22);
                        hipEventRecord(stop22,0);
                        hipEventSynchronize(stop22);
                        hipEventElapsedTime(&time22, start22, stop22);
			time22 = time22 + time22;
			hipDeviceSynchronize();
			hipMemcpy(h_A, d_A, sizeof(int) * 1, hipMemcpyDeviceToHost);
			tempMax = h_A[0];
			}
			}
			k++;
			time2 = time2 + time2;
			}	
			time29 = (time2 + time22 + time21) / 10;
                       // time29 = time29/10;
                        time29 = time29/1000; 
                        m1 = tempMax;	  
		        gpu1 = time29;
        helper(h_A, N);
	hipFree(d_A);
	free(h_A);
cc1 = cpu1 / gpu1;
}

void step1Min (int N){
hipEvent_t start3, stop3;
hipEvent_t start31, stop31;
hipEvent_t start32, stop32;
float time3;
float time32;
float time31;
N = N * 1048576;
        

    

	float *d_B;
    int i;
    size_t size = N *sizeof(float);
    float *h_B = (float *)malloc(size);


	hipMalloc( (void **)&d_B, sizeof(float) * N);
	

	for(i = 0; i < N; i++)
	{
		h_B[i] = (float)rand();
	}

		
	float tempMin;
                
		

		        int blockGridWidth = BLOCKS_PER_GRID_ROW;
		        int blockGridHeight = (N / THREADS_PER_BLOCK) / blockGridWidth;

		        dim3 blockGridRows(blockGridWidth, blockGridHeight);
		        dim3 threadBlockRows(THREADS_PER_BLOCK, 1);
                        int k=0;
			while (k!=10)
			{		
                        hipMemcpy(d_B, h_B, sizeof(float) * N, hipMemcpyHostToDevice);
                        hipEventCreate(&start3);
                        hipEventRecord(start3,0);
			erredd<<<blockGridRows, threadBlockRows>>>(d_B);
			hipEventCreate(&stop3);
                        hipEventRecord(stop3,0);
                        hipEventSynchronize(stop3);
                        hipEventElapsedTime(&time3, start3, stop3);
			hipDeviceSynchronize();
			hipMemcpy(h_B, d_B, sizeof(float) * N / THREADS_PER_BLOCK, hipMemcpyDeviceToHost);
			tempMin = h_B[0];
			k++;
			time3 = time3 + time3;
			for (int i = N / THREADS_PER_BLOCK; i > 0; i = i / 2)
			{
			hipMemcpy(d_B, h_B, sizeof(float) * i, hipMemcpyHostToDevice);
                        hipEventCreate(&start31);
                        hipEventRecord(start31,0);
			erredd<<<blockGridRows, threadBlockRows>>>(d_B);
			hipEventCreate(&stop31);
                        hipEventRecord(stop31,0);
                        hipEventSynchronize(stop31);
                        hipEventElapsedTime(&time31, start31, stop31);
			hipDeviceSynchronize();
			time31 = time31 + time31;
			hipMemcpy(h_B, d_B, sizeof(float) * i, hipMemcpyDeviceToHost);
			tempMin = h_B[0];
			if (i==1)
			{
                        hipMemcpy(d_B, h_B, sizeof(int) * THREADS_PER_BLOCK, hipMemcpyHostToDevice);
                        hipEventCreate(&start32);
                        hipEventRecord(start32,0);
			erredd<<<blockGridRows, threadBlockRows>>>(d_B);
			hipEventCreate(&stop32);
                        hipEventRecord(stop32,0);
                        hipEventSynchronize(stop32);
                        hipEventElapsedTime(&time32, start32, stop32);
			hipDeviceSynchronize();
			time32 = time32 + time32;
			hipMemcpy(h_B, d_B, sizeof(int) * 1, hipMemcpyDeviceToHost);
			tempMin = h_B[0];
			}
			}
			}	        
			tempMin = h_B[0];			
		gpu2 = (time31+time3 + time32) / 10;	
   //               gpu2 = gpu2 / 100;
                gpu2 = gpu2 / 1000;
		m2 = tempMin;
		helper2(h_B, N);	
	hipFree(d_B);
	free(h_B);

cc2 = cpu2 / gpu2;
}

int main(int argc, char **argv){
int a[3] = {2, 8, 32};
float element1;
printf("Step 1\n");
printf("Shuyang\n");
printf("Zang\n");
//printf("N   2M   GPUmax   %f   CPUmax  %f   GPUtime  %f   CPUtime  %f  GPUSpeedup \n");
for (int i=0; i<3;i++){
step1Max(a[i]);
element1 = a[i];
printf("N   %f   GPUmax   %f   CPUmax   %f   GPUtime   %f   CPUtime   %f   GPUSpeedup   %f \n", element1, m1, max1, gpu1, cpu1, cc1);
//printf("%6f   ", element1);
//printf("%12f   ", m1);
//printf("%12f   ", max1);
//printf("%12f   ", gpu1);
//printf("%16f   ", cpu1);
//printf("%25f   \n", cc1);
}
printf("\n");
//printf("N            GPUmin                    CPUmin           GPUtime           CPUtime                    GPUSpeedup \n");
for (int i=0; i<3;i++){
step1Min(a[i]);
element1 = a[i];
printf("N   %f   GPUmax   %f   CPUmax   %f   GPUtime   %f   CPUtime   %f   GPUSpeedup   %f \n", element1, m2, min1, gpu2, cpu2, cc2);
//printf("%6f   ", element1);
//printf("%16f   ", m2);
//printf("%16f   ", min1);
//printf("%16f   ", gpu2);
//printf("%20f   ", cpu2);
//printf("%24f   \n", cc2);
}
}
